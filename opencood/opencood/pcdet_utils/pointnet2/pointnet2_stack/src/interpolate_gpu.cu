#include "hip/hip_runtime.h"
/*
Stacked-batch-data version of point interpolation, modified from the original implementation of official PointNet++ codes.
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/


#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "interpolate_gpu.h"


__global__ void three_nn_kernel_stack(int batch_size, int N, int M, const float *unknown, 
    const int *unknown_batch_cnt, const float *known, const int *known_batch_cnt,
    float *dist2, int *idx) {
    // unknown: (N1 + N2 ..., 3)
    // unknown_batch_cnt: (batch_size), [N1, N2, ...]
    // known: (M1 + M2 ..., 3)
    // known_batch_cnt: (batch_size), [M1, M2, ...]
    // Return:
    // dist: (N1 + N2 ..., 3)  l2 distance to the three nearest neighbors
    // idx: (N1 + N2 ..., 3)  index of the three nearest neighbors

    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= N) return;

    int bs_idx = 0, pt_cnt = unknown_batch_cnt[0];
    for (int k = 1; k < batch_size; k++){
        if (pt_idx < pt_cnt) break;
        pt_cnt += unknown_batch_cnt[k];
        bs_idx = k;
    }

    int cur_num_known_points = known_batch_cnt[bs_idx];

    int known_batch_start_idx = 0;
    for (int k = 0; k < bs_idx; k++) known_batch_start_idx += known_batch_cnt[k];

    known += known_batch_start_idx * 3;
    unknown += pt_idx * 3;
    dist2 += pt_idx * 3;
    idx += pt_idx * 3;

    float ux = unknown[0];
    float uy = unknown[1];
    float uz = unknown[2];

    double best1 = 1e40, best2 = 1e40, best3 = 1e40;
    int besti1 = 0, besti2 = 0, besti3 = 0;
    for (int k = 0; k < cur_num_known_points; ++k) {
        float x = known[k * 3 + 0];
        float y = known[k * 3 + 1];
        float z = known[k * 3 + 2];
        float d = (ux - x) * (ux - x) + (uy - y) * (uy - y) + (uz - z) * (uz - z);
        if (d < best1) {
            best3 = best2; besti3 = besti2;
            best2 = best1; besti2 = besti1;
            best1 = d; besti1 = k;
        } 
        else if (d < best2) {
            best3 = best2; besti3 = besti2;
            best2 = d; besti2 = k;
        } 
        else if (d < best3) {
            best3 = d; besti3 = k;
        }
    }
    dist2[0] = best1; dist2[1] = best2; dist2[2] = best3;
    idx[0] = besti1 + known_batch_start_idx; 
    idx[1] = besti2 + known_batch_start_idx; 
    idx[2] = besti3 + known_batch_start_idx;
}


void three_nn_kernel_launcher_stack(int batch_size, int N, int M, const float *unknown, 
    const int *unknown_batch_cnt, const float *known, const int *known_batch_cnt,
    float *dist2, int *idx) {
    // unknown: (N1 + N2 ..., 3)
    // unknown_batch_cnt: (batch_size), [N1, N2, ...]
    // known: (M1 + M2 ..., 3)
    // known_batch_cnt: (batch_size), [M1, M2, ...]
    // Return:
    // dist: (N1 + N2 ..., 3)  l2 distance to the three nearest neighbors
    // idx: (N1 + N2 ..., 3)  index of the three nearest neighbors

    hipError_t err;
    dim3 blocks(DIVUP(N, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    three_nn_kernel_stack<<<blocks, threads>>>(
        batch_size, N, M, unknown, unknown_batch_cnt, 
        known, known_batch_cnt, dist2, idx
    );

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}



__global__ void three_interpolate_kernel_stack(int N, int channels, const float *features, 
    const int *idx, const float *weight, float *out) {
    // features: (M1 + M2 ..., C)
    // idx: [N1 + N2 ..., 3]
    // weight: [N1 + N2 ..., 3]
    // Return:
    // out: (N1 + N2 ..., C)

    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= N || c_idx >= channels) return;

    weight += pt_idx * 3;
    idx += pt_idx * 3;
    out += pt_idx * channels + c_idx;

    out[0] = weight[0] * features[idx[0] * channels + c_idx] + 
        weight[1] * features[idx[1] * channels + c_idx] + 
        weight[2] * features[idx[2] * channels + c_idx];
}



void three_interpolate_kernel_launcher_stack(int N, int channels,
    const float *features, const int *idx, const float *weight, float *out) {
    // features: (M1 + M2 ..., C)
    // idx: [N1 + N2 ..., 3]
    // weight: [N1 + N2 ..., 3]
    // Return:
    // out: (N1 + N2 ..., C)

    hipError_t err;
    dim3 blocks(DIVUP(N, THREADS_PER_BLOCK), channels);
    dim3 threads(THREADS_PER_BLOCK);
    three_interpolate_kernel_stack<<<blocks, threads>>>(N, channels, features, idx, weight, out);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void three_interpolate_grad_kernel_stack(int N, int channels, const float *grad_out, 
    const int *idx, const float *weight, float *grad_features) {
    // grad_out_tensor: (N1 + N2 ..., C)
    // idx_tensor: [N1 + N2 ..., 3]
    // weight_tensor: [N1 + N2 ..., 3]
    // Return:
    // grad_features_tensor: (M1 + M2 ..., C)

    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= N || c_idx >= channels) return;

    grad_out += pt_idx * channels + c_idx;
    weight += pt_idx * 3;
    idx += pt_idx * 3;
    
    // printf("pt_idx=%d, c_idx=%d, idx=(%d, %d, %d), grad_out=%f\n", pt_idx, c_idx, idx[0], idx[1], idx[2], grad_out[0]);

    atomicAdd(grad_features + idx[0] * channels + c_idx, grad_out[0] * weight[0]);
    atomicAdd(grad_features + idx[1] * channels + c_idx, grad_out[0] * weight[1]);
    atomicAdd(grad_features + idx[2] * channels + c_idx, grad_out[0] * weight[2]);
}


void three_interpolate_grad_kernel_launcher_stack(int N, int channels, const float *grad_out, 
    const int *idx, const float *weight, float *grad_features) {
    // grad_out_tensor: (N1 + N2 ..., C)
    // idx_tensor: [N1 + N2 ..., 3]
    // weight_tensor: [N1 + N2 ..., 3]
    // Return:
    // grad_features_tensor: (M1 + M2 ..., C)

    hipError_t err;
    dim3 blocks(DIVUP(N, THREADS_PER_BLOCK), channels);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);
    three_interpolate_grad_kernel_stack<<<blocks, threads>>>(
        N, channels, grad_out, idx, weight, grad_features
    );

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}